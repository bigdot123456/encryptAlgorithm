#include "hip/hip_runtime.h"
/*
 �����ܳ���ʵ�ֶ��ļ��Ľ���
*/
#include "md5.h"
#include "AES.h"


int runDeCry(char* md5key,unsigned char * Imem,unsigned char * Omem,unsigned long mem_length); ////����
unsigned long GetFileLen1(const char* szFilePath); //�õ��ļ��ĳ���

extern "C" 
int jiemi(char* md5key,char* filepath) 
{
	int deviceCount = 0;
	CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount));
	
	//û��֧��CUDA���豸
	if(deviceCount==0)
	{
		printf("�����豸��֧��CUDA��\n");
		return -1;
	}

	//���������汾
    int dev;
	int driverVersion = 0;     
    for (dev = 0; dev < deviceCount; ++dev) 
	{
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
		if(CUDART_VERSION >= 2020)
		{
			hipDriverGetVersion(&driverVersion);
			//printf("CUDA Driver Version: %d.%d\n", driverVersion/1000, driverVersion%100);
			if(driverVersion/1000 < 3 || driverVersion/1000 == 3 && driverVersion%100 < 2)
			{
				printf("�����Կ������汾̫�ͣ�������Կ�������\n");
				return -2;
			}
		}
		else
			printf("�����Կ������汾̫�ͣ�������Կ�������\n");
	}

	//��������ʱ��
	clock_t start, finish, cost;
	double totaltime;

	FILE *fp;							//���ļ��ж�������
	if((fp=fopen(filepath,"rb"))==NULL)
	{
		printf("�޷�������ѡ����ļ�\n");
		exit(0);
	}
	unsigned long input_length = GetFileLen1(filepath);	        //���볤��
	unsigned long mem_length = (input_length + 1024 * 16 - 1) / 4;	//�洢������,��16k����
	unsigned char *Aes;									            //�ڴ��е�����
	Aes = (unsigned char*) malloc(sizeof(unsigned int) * mem_length);	//���ڴ���Ϊ���ķ���ռ�
	unsigned char *OAes;									               //�ڴ��е�����
	OAes = (unsigned char*) malloc(sizeof(unsigned int) * mem_length);  //���ڴ���Ϊ���ķ���ռ�
	
	fread(Aes, sizeof(unsigned char), input_length, fp);
	for(unsigned int i = input_length; i < 4 * mem_length; i ++)
	{
		Aes[i] = 0;
	}
	fclose(fp);

	printf("\n");
	printf("���ڽ���, AES128, EBC mode ...\n");

	start=clock();
    runDeCry(md5key,Aes,OAes,mem_length);
	finish=clock();

	char filename[260];
	strcpy(filename,filepath);
	strcat(filename,".changeme");
//	int size;
	//filename[strlen(filename) - 4] = '\0';			//�����ļ���Ϊ�����ļ�ȥ����׺.bfe
	FILE* fp_w = fopen(filename,"wb");
//	printf("%s",filename);
	//д������ļ�
	fwrite(OAes, sizeof(unsigned char), (input_length + 15) / 16 * 16, fp_w);
//	printf("%d\n",size);

	fclose(fp_w);
		
	// �ͷſռ�
	free(Aes);
	free(OAes);

	totaltime=(double)(finish-start)/CLOCKS_PER_SEC;
    printf("��������ʱ��Ϊ%f��! �������ݴ����ٶ�%f MBytes/s!\n",totaltime, input_length / totaltime / 1024 / 1024);

	return 0;
}

//����
int runDeCry(char* md5key,unsigned char * Imem,unsigned char * Omem,unsigned long mem_length) 
{
	unsigned char *IAes;
	unsigned char *OAes;
	int nouse;
	unsigned char mykey[16]; //��չ���� 
	int round;
	//MD5��չ�û�����
	MD5 md5;
/*	if(argc < 2)		//���û�������ļ��������˳���
	{
		printf("��ѡ��Ҫ���ܵ��ļ��������������룡\n");
		return -1;
	}
*/	
    md5.Data((unsigned char *)md5key,strlen(md5key),mykey);

	//����Ҫ���ܵ��ļ�
//	for(int k = 2; k < argc; ++k)
//	{

		//����û����������Ƿ���ȷ
/*		unsigned char pswd[16];
		fread(pswd, sizeof(unsigned char), 16, fp);
		if(memcmp(pswd,mykey,sizeof(pswd)))
		{
			printf("�û�����У��ʧ�ܣ�\n");
			return 2;
		}
*/
	unsigned int *roundkey;									            //�ڴ��е�����
	roundkey = (unsigned int*) malloc(sizeof(unsigned int) * 44);	    //���ڴ���Ϊ���ķ���ռ�
	if(mem_length < PIECE_SIZE)
	{
		unsigned int* d_roundkey;
		
		CUDA_SAFE_CALL( hipMalloc( (void**) &d_roundkey, sizeof(unsigned int) * 44 ));
		
		AesSetKeyDecode(roundkey, mykey, 16);//�����������Կ
		
		CUDA_SAFE_CALL( hipMemcpy( d_roundkey, roundkey, sizeof(unsigned int) * 44 ,hipMemcpyHostToDevice) );	//������Կ�������Դ���	
		//��ʼ��CUDA���л���
	//	hipSetDevice(0);	

		//Ϊ���ķ����Դ�
		unsigned int* d_Aes;		
		CUDA_SAFE_CALL( hipMalloc( (void**) &d_Aes, sizeof(unsigned int) * (mem_length )));

		//Ϊ��������Դ�
		unsigned int* d_OAes;			
		CUDA_SAFE_CALL( hipMalloc( (void**) &d_OAes, sizeof(unsigned int) * mem_length ));

	
		//�����Ŀ������Դ���
		CUDA_SAFE_CALL( hipMemcpy( d_Aes, Imem, sizeof(unsigned int) * mem_length, hipMemcpyHostToDevice) );

		// ����ִ�в���
		dim3  grid( (mem_length ) / BLOCK_SIZE / LOOP_IN_BLOCK , 1, 1);		//����grid, grid��СΪ ���ĳ���/ һ��BLOCK�д����32bit integer�� / BLOCK��ѭ������												
		dim3  threads( BLOCK_SIZE, 1, 1);

		/*��ʼ����*/							
		AES128_EBC_decry_kernel<<< grid, threads>>>(d_Aes, d_OAes, d_roundkey);

		CUT_CHECK_ERROR("Kernel execution failed\n");	//����Ƿ���ȷִ��

		CUDA_SAFE_CALL( hipMemcpy( Omem, d_OAes, sizeof(unsigned int) * mem_length,hipMemcpyDeviceToHost) );//��������Դ濽�����ڴ�
			
		//��������ļ����ļ���
		free(roundkey);

		CUDA_SAFE_CALL(hipFree(d_Aes));
		CUDA_SAFE_CALL(hipFree(d_OAes));
		CUDA_SAFE_CALL(hipFree(d_roundkey));
	}
	else
	{
		printf("->\n");
		IAes = Imem;
		OAes = Omem;
		unsigned long mem_remainder;
		int time;
		round = mem_length/PIECE_SIZE;
		mem_remainder = mem_length%PIECE_SIZE;


		unsigned int* d_roundkey;
		CUDA_SAFE_CALL( hipMalloc( (void**) &d_roundkey, sizeof(unsigned int) * 44 ));
		
		AesSetKeyDecode(roundkey, mykey, 16);//�����������Կ
		
		CUDA_SAFE_CALL( hipMemcpy( d_roundkey, roundkey, sizeof(unsigned int) * 44 ,hipMemcpyHostToDevice) );	//������Կ�������Դ���	
		//��ʼ��CUDA���л���
	//	hipSetDevice(0);	
  		printf("���ڽ���...\n");
		unsigned int* d_Aes;		

		unsigned int* d_OAes;			

		//Ϊ���ķ����Դ�
		CUDA_SAFE_CALL( hipMalloc( (void**) &d_Aes, sizeof(unsigned int) * PIECE_SIZE));

		//Ϊ��������Դ�
		CUDA_SAFE_CALL( hipMalloc( (void**) &d_OAes, sizeof(unsigned int) * PIECE_SIZE ));

			
		for(time = 0;time < round;time++)
		{
 //			printf("%d\n",IAes);
//			printf("%d\n",OAes);


			//�����Ŀ������Դ���
			CUDA_SAFE_CALL( hipMemcpy( d_Aes, IAes, sizeof(unsigned int) * PIECE_SIZE, hipMemcpyHostToDevice) );

			// ����ִ�в���
			dim3  grid( PIECE_SIZE / BLOCK_SIZE / LOOP_IN_BLOCK , 1, 1);		//����grid, grid��СΪ ���ĳ���/ һ��BLOCK�д����32bit integer�� / BLOCK��ѭ������												
			dim3  threads( BLOCK_SIZE, 1, 1);

			/*��ʼ����*/

			AES128_EBC_decry_kernel<<< grid, threads>>>(d_Aes, d_OAes, d_roundkey);

			CUT_CHECK_ERROR("Kernel execution failed\n");	//����Ƿ���ȷִ��

			CUDA_SAFE_CALL( hipMemcpy( OAes, d_OAes, sizeof(unsigned int) * PIECE_SIZE,hipMemcpyDeviceToHost) );//��������Դ濽�����ڴ�
			IAes = Imem + (time + 1)*PIECE_SIZE*4;
			OAes = Omem + (time + 1)*PIECE_SIZE*4;


		}
		//��������ļ����ļ���

		CUDA_SAFE_CALL(hipFree(d_Aes));
		CUDA_SAFE_CALL(hipFree(d_OAes));

		CUDA_SAFE_CALL( hipMalloc( (void**) &d_Aes, sizeof(unsigned int) * (mem_remainder)));
		CUDA_SAFE_CALL( hipMalloc( (void**) &d_OAes, sizeof(unsigned int) * (mem_remainder)));
		CUDA_SAFE_CALL( hipMemcpy( d_Aes, IAes, sizeof(unsigned int) * (mem_remainder), hipMemcpyHostToDevice) );

		// �������в���
		dim3  grid( (mem_remainder) / BLOCK_SIZE / LOOP_IN_BLOCK , 1, 1);		//����grid, grid��СΪ ���ĳ���/ һ��BLOCK�д����32bit integer�� / BLOCK��ѭ������												
		dim3  threads( BLOCK_SIZE, 1, 1);

		AES128_EBC_decry_kernel<<< grid, threads>>>(d_Aes, d_OAes, d_roundkey);

		CUT_CHECK_ERROR("CUDA�ں�ִ��ʧ�ܣ�\n");	//����Ƿ���ȷִ��


		CUDA_SAFE_CALL( hipMemcpy( OAes, d_OAes, sizeof(unsigned int) * mem_remainder,hipMemcpyDeviceToHost) );//��������Դ濽�����ڴ�	

		free(roundkey);
		CUDA_SAFE_CALL(hipFree(d_Aes));
		CUDA_SAFE_CALL(hipFree(d_OAes));
		CUDA_SAFE_CALL(hipFree(d_roundkey));
	}

//	}	
	return 0;
}
	
unsigned long GetFileLen1(const char* szFilePath)
{
	FILE* pFile = fopen(szFilePath, "rb");
	if (pFile == NULL)
		return -1;

	fseek(pFile, 0, SEEK_END);
	long nFileLen = ftell(pFile);
	fclose(pFile);

	return nFileLen;
}
