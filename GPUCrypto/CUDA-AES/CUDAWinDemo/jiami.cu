#include "hip/hip_runtime.h"
 /*
  ���ܳ���ʵ�ֶ��ļ��ļ���
*/

#include "md5.h"
#include "AES.h"
#include "stdio.h"

//run AES test
int runCry(char* md5key,unsigned char * Imem,unsigned char * Omem,unsigned long mem_length); 
unsigned long GetFileLen(const char* szFilePath); //�õ��ļ��ĳ���

extern "C" 
int jiami(char* md5key,char* filepath) 
{
	int deviceCount = 0;
	CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount));
	
	//û��֧��CUDA���豸
	if(deviceCount==0)
	{
		printf("�����豸��֧��CUDA��\n");
		return -1;
	}

	//���������汾
    int dev;
	int driverVersion = 0;     
    for (dev = 0; dev < deviceCount; ++dev) 
	{
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
		if(CUDART_VERSION >= 2020)
		{
			hipDriverGetVersion(&driverVersion);
			//printf("CUDA Driver Version: %d.%d\n", driverVersion/1000, driverVersion%100);
			if(driverVersion/1000 < 3 || driverVersion/1000 == 3 && driverVersion%100 < 2)
			{
				printf("�����Կ������汾̫�ͣ�������Կ�������\n");
				return -2;
			}
		}
		else
		{
			printf("�����Կ������汾̫�ͣ�������Կ�������\n");
			return -2;
		}
	}

	//��������ʱ��
	clock_t start, finish, cost;
	double totaltime;
	FILE *fp;   //���ļ��ж�������											
	
	if((fp=fopen(filepath,"rb"))==NULL)
	{
		printf("�޷�������ѡ����ļ�\n");
		exit(0);
	}

	unsigned long input_length = GetFileLen( filepath );	               //�ļ�����
	unsigned long mem_length = (input_length + 1024 * 16 - 1) / 4;	   //�洢������,��16k����
	unsigned char *Aes;									               //�ڴ��е�����
	Aes = (unsigned char*) malloc(sizeof(unsigned int) * mem_length);  //���ڴ���Ϊ���ķ���ռ�
	unsigned char *OAes;									               //�ڴ��е�����
	OAes = (unsigned char*) malloc(sizeof(unsigned int) * mem_length);  //���ڴ���Ϊ���ķ���ռ�
	
	//�������ļ������ڴ�
	fread(Aes, sizeof(unsigned char), input_length, fp);
	for(unsigned int i = input_length; i < 4 * mem_length; i ++)
	{
		Aes[i] = 0;
	}
	fclose(fp);

	//�õ��ڴ�������ݵ�ʱ��
	start=clock();
	runCry(md5key,Aes,OAes,mem_length);
	finish=clock();

	cost = finish - start;
	totaltime = (double)cost / CLOCKS_PER_SEC;
    
	printf("��������ʱ��Ϊ%f��! �������ݴ����ٶ�%f MBytes/s!\n",totaltime, input_length / totaltime / 1024 / 1024);

	char filename[260];
	strcpy(filename,filepath);	
	strcat(filename,".enc");
//	printf("%s",filename);
	//д������ļ�
	FILE* fp_w = fopen(filename,"wb");
	if(fp_w == NULL)
	{
		printf("���ļ�ʧ��");
	}
//	int size;
//	size = fwrite(mykey,sizeof(unsigned char),16,fp_w);	//���û�MD5����д���ļ�ͷ
//	printf("%d\n",size);
	fwrite(OAes, sizeof(unsigned char), (input_length + 15) / 16 * 16, fp_w);
//	printf("%d\n",size);
	fclose(fp_w);

	// �ͷſռ�
	free(Aes);
	free(OAes);

	finish=clock();
	totaltime=(double)(finish-start)/CLOCKS_PER_SEC;
//    printf("\n���м�д�ļ�ʱ��Ϊ%f��!\n",totaltime);

	return 0;
}

// ����
int runCry(char* md5key,unsigned char * Imem,unsigned char * Omem,unsigned long mem_length) 
{
	unsigned char *IAes;
	unsigned char *OAes;
	unsigned char mykey[16]; //��չ���� 
	int round;
	//MD5��չ�û�����
	MD5 md5;
/*	if(argc < 2)		//���û�������ļ��������˳���
	{
		printf("��ѡ��Ҫ���ܵ��ļ��������������룡\n");
		return -1;
	}
*/
	md5.Data((unsigned char *)md5key,strlen(md5key),mykey);

   //hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

	//����Ҫ���ܵ��ļ�
//	for(int k = 2; k < argc; ++k)
//	{
	unsigned int *roundkey;							                   //�ڴ��е�����
	roundkey = (unsigned int*) malloc(sizeof(unsigned int) * 44);	   //���ڴ���Ϊ���ķ���ռ�
	
	printf("mem_length: %d\n",mem_length);
	printf("PIECE_SIZE %d\n",PIECE_SIZE);
	
	//���ݳ���<16M
	if (mem_length < PIECE_SIZE)
	//��ʼ��CUDA���л���
	{
		hipSetDevice(0);	

		unsigned int* d_roundkey;
		CUDA_SAFE_CALL( hipMalloc( (void**) &d_roundkey, sizeof(unsigned int) * 44 ));

		AesSetKeyEncode(roundkey, mykey, 16);//�����������Կ,128bit��Կ������16Byte

		CUDA_SAFE_CALL( hipMemcpy( d_roundkey, roundkey, sizeof(unsigned int) * 44 ,hipMemcpyHostToDevice) );	//������Կ�������Դ���	


		//Ϊ���ķ����Դ�
		unsigned int* d_Aes;		
		CUDA_SAFE_CALL( hipMalloc( (void**) &d_Aes, sizeof(unsigned int) * (mem_length )));

		//Ϊ��������Դ�
		unsigned int* d_OAes;			
		CUDA_SAFE_CALL( hipMalloc( (void**) &d_OAes, sizeof(unsigned int) * mem_length ));

		//�����Ŀ������Դ���
		CUDA_SAFE_CALL( hipMemcpy( d_Aes, Imem, sizeof(unsigned int) * mem_length, hipMemcpyHostToDevice) );

		// �������в���
		// grid�еĵ�һ���ڶ������������65535�� �����������㶨Ϊ1.
		dim3  grid( (mem_length ) / BLOCK_SIZE / LOOP_IN_BLOCK , 1, 1);		//����grid, grid��СΪ ���ĳ���/ һ��BLOCK�д����32bit integer�� / BLOCK��ѭ������												
		dim3  threads( BLOCK_SIZE, 1, 1);

		/*���ܿ�ʼ*/
		printf("\n");
		printf("���ڼ���, AES128, EBC mode ...\n");
		
		//d_Aes���Դ��е����������ַ
		//d_OAes���Դ��е����������ַ

		AES128_EBC_encry_kernel<<<grid, threads>>>(d_Aes, d_OAes, d_roundkey); //���ܳ����ں�

		CUT_CHECK_ERROR("CUDA�ں�ִ��ʧ�ܣ�\n");	//����Ƿ���ȷִ��

		CUDA_SAFE_CALL( hipMemcpy( Omem, d_OAes, sizeof(unsigned int) * mem_length,hipMemcpyDeviceToHost) );//��������Դ濽�����ڴ�	
			
		//��������ļ����ļ���
		free(roundkey);

		CUDA_SAFE_CALL(hipFree(d_Aes));
		CUDA_SAFE_CALL(hipFree(d_OAes));
		CUDA_SAFE_CALL(hipFree(d_roundkey));
	}
	else
	{
		printf("+>\n");

		IAes = Imem;
		OAes = Omem;
		unsigned long mem_remainder;
		int time;
		round = mem_length/PIECE_SIZE;		 //64M����
		mem_remainder = mem_length%PIECE_SIZE;
		printf("���ڼ���...\n");
		hipSetDevice(0);	

		unsigned int* d_roundkey;
		CUDA_SAFE_CALL( hipMalloc( (void**) &d_roundkey, sizeof(unsigned int) * 44 ));


		AesSetKeyEncode(roundkey, mykey, 16);//�����������Կ,128bit��Կ������16Byte

		CUDA_SAFE_CALL( hipMemcpy( d_roundkey, roundkey, sizeof(unsigned int) * 44 ,hipMemcpyHostToDevice) );	//������Կ�������Դ���	
		unsigned int* d_Aes;		

		unsigned int* d_OAes;			

		//Ϊ���ķ����Դ�
		CUDA_SAFE_CALL( hipMalloc( (void**) &d_Aes, sizeof(unsigned int) * PIECE_SIZE));

		//Ϊ��������Դ�
		CUDA_SAFE_CALL( hipMalloc( (void**) &d_OAes, sizeof(unsigned int) * PIECE_SIZE));
		printf("%d\n",round);
		for(time = 0;time < round;time++)
		{

			//�����Ŀ������Դ���
			CUDA_SAFE_CALL( hipMemcpy( d_Aes, IAes, sizeof(unsigned int) * PIECE_SIZE, hipMemcpyHostToDevice) );

			// �������в���
			dim3  grid( PIECE_SIZE / BLOCK_SIZE / LOOP_IN_BLOCK , 1, 1);		//����grid, grid��СΪ ���ĳ���/ һ��BLOCK�д����32bit integer�� / BLOCK��ѭ������												
			dim3  threads( BLOCK_SIZE, 1, 1);

			/*���ܿ�ʼ*/
									
			AES128_EBC_encry_kernel<<< grid, threads>>>(d_Aes, d_OAes, d_roundkey); //���ܳ����ں�
//			Sleep(5000);
			CUT_CHECK_ERROR("CUDA�ں�ִ��ʧ�ܣ�\n");	//����Ƿ���ȷִ��

			printf("%d\n",IAes);
			printf("%d\n",OAes);
			CUDA_SAFE_CALL( hipMemcpy( OAes, d_OAes, sizeof(unsigned int) * PIECE_SIZE,hipMemcpyDeviceToHost) );//��������Դ濽�����ڴ�	
			IAes = Imem + (time + 1)*PIECE_SIZE*4;
			OAes = Omem + (time + 1)*PIECE_SIZE*4;

		}	
		//��������ļ����ļ���
//		CUDA_SAFE_CALL(hipFree(d_Aes));
//		CUDA_SAFE_CALL(hipFree(d_OAes));



//		CUDA_SAFE_CALL( hipMalloc( (void**) &d_Aes, sizeof(unsigned int) * (mem_remainder)));
//		CUDA_SAFE_CALL( hipMalloc( (void**) &d_OAes, sizeof(unsigned int) * (mem_remainder)));
		CUDA_SAFE_CALL( hipMemcpy( d_Aes, IAes, sizeof(unsigned int) * (mem_remainder), hipMemcpyHostToDevice) );
		printf("%d\n",mem_remainder);
		// �������в���
		dim3  grid( (mem_remainder) / BLOCK_SIZE / LOOP_IN_BLOCK , 1, 1);		//����grid, grid��СΪ ���ĳ���/ һ��BLOCK�д����32bit integer�� / BLOCK��ѭ������												
		dim3  threads( BLOCK_SIZE, 1, 1);

		/*���ܿ�ʼ*/
								
		AES128_EBC_encry_kernel<<< grid, threads>>>(d_Aes, d_OAes, d_roundkey); //���ܳ����ں�

		CUT_CHECK_ERROR("CUDA�ں�ִ��ʧ�ܣ�\n");	//����Ƿ���ȷִ��


		CUDA_SAFE_CALL( hipMemcpy( OAes, d_OAes, sizeof(unsigned int) * mem_remainder,hipMemcpyDeviceToHost) );//��������Դ濽�����ڴ�	

		free(roundkey);
		CUDA_SAFE_CALL(hipFree(d_Aes));
		CUDA_SAFE_CALL(hipFree(d_OAes));

		CUDA_SAFE_CALL(hipFree(d_roundkey));	
	}
//	}	
	return 0;
}
	
unsigned long GetFileLen(const char* szFilePath)  //�õ��ļ��ĳ���
{
	FILE* pFile = fopen(szFilePath, "rb");
	if (pFile == NULL)
		return -1;

	fseek(pFile, 0, SEEK_END);
	long nFileLen = ftell(pFile);
	fclose(pFile);

	return nFileLen;
}
